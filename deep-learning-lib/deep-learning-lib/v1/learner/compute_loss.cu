/* Copyright 2016, 2019 S�bastien K�roack. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
==============================================================================*/

#include "deep-learning-lib/v1/learner/model.cuh"

__device__ void cuModel::Test(size_t const batch_size,
                                                          var **const ptr_array_outputs_received,
                                                          size_t const time_step_index_received)
{ this->FF__Test(batch_size, ptr_array_outputs_received); }

__device__ void cuModel::FF__Test(size_t const batch_size, var **const ptr_array_outputs_received)
{
    switch(this->type_loss_function)
    {
        case DL::LOSS_FN::ME:
        case DL::LOSS_FN::L1:
        case DL::LOSS_FN::MAE:
        case DL::LOSS_FN::L2:
        case DL::LOSS_FN::MSE:
        case DL::LOSS_FN::RMSE:
        case DL::LOSS_FN::MAPE:
        case DL::LOSS_FN::SMAPE:
        case DL::LOSS_FN::MASE_SEASONAL:
        case DL::LOSS_FN::MASE_NON_SEASONAL:
            this->FF__Test__Standard(batch_size, ptr_array_outputs_received);
                break;
        case DL::LOSS_FN::CROSS_ENTROPY:
            this->FF__Test__Binary_Cross_Entropy(batch_size, ptr_array_outputs_received);
                break;
        case DL::LOSS_FN::BIT: this->FF__Test__Bit_Fail(batch_size, ptr_array_outputs_received); break;
        default:
            ERR(L"Undefined type loss function (%u).",
                                    this->type_loss_function);
                break;
    }
}

__device__ void cuModel::RNN__Test(size_t const batch_size,
                                                                    var **const ptr_array_outputs_received,
                                                                    size_t const time_step_index_received)
{
}
