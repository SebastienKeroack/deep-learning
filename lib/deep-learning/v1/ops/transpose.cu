#include "hip/hip_runtime.h"
﻿/* Copyright 2016, 2019 Sébastien Kéroack. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
==============================================================================*/

#include "deep-learning/v1/data/shared_memory.cuh"

namespace DL::v1::Math {
// TODO: Fix. Some dimension like 25x25 have bank conflict.
template <typename T, size_t TILE_WIDTH>
__global__ void kernel__Transpose_Square(
    size_t const size_received, size_t const width_received,
    T *const ptr_array_outputs_received,
    T const *const ptr_array_inputs_received) {
  size_t const &tmp_thread_block_index_x(threadIdx.x),
      &tmp_thread_block_index_y(threadIdx.y),
      tmp_thread_block_diagonal(
          (tmp_thread_block_index_x + tmp_thread_block_index_y) % TILE_WIDTH),
      tmp_grid_stride_x(gridDim.x * TILE_WIDTH * 2u),
      tmp_grid_stride_y(gridDim.y * TILE_WIDTH * 2u);
  size_t tmp_thread_global_index_block_x(blockIdx.x * TILE_WIDTH * 2u),
      tmp_thread_global_index_block_y(blockIdx.y * TILE_WIDTH * 2u),
      tmp_thread_global_index_x, tmp_thread_global_index_y,
      tmp_thread_global_index_offSet__x, tmp_thread_global_index_offSet__y;

  __shared__ T tmp_array_tile[TILE_WIDTH * 4u][TILE_WIDTH];

  while (tmp_thread_global_index_block_y < width_received) {
    while (tmp_thread_global_index_block_x < width_received) {
      // Coalesced index X.
      // 0 * 32 * 2 + [0...1...31] = 0 + [0...1...31]
      // 1 * 32 * 2 + [0...1...31] = 64 + [0...1...31]
      tmp_thread_global_index_x =
          tmp_thread_global_index_block_x + tmp_thread_block_index_x;
      tmp_thread_global_index_offSet__x =
          tmp_thread_global_index_x + TILE_WIDTH;

      // Coalesced index Y.
      // 0 * 32 * 2 + [0...1...31] = 0 + [0...1...31]
      // 1 * 32 * 2 + [0...1...31] = 64 + [0...1...31]
      tmp_thread_global_index_y =
          tmp_thread_global_index_block_y + tmp_thread_block_index_y;
      tmp_thread_global_index_offSet__y =
          tmp_thread_global_index_y + TILE_WIDTH;

      if (tmp_thread_global_index_offSet__x < width_received &&
          tmp_thread_global_index_offSet__y < width_received) {
        // Coalesced Shared Memory:
        // Y[0]: [0][0], [0][1], [0][2], [0][3], [0][4], [0][5], [0][6], ...,
        // [Y][X] = ... Coalesced Global Memory: Y[0]: ... = [0 * 8 + 0 == 000],
        // [0 * 8 + 1 == 001], [0 * 8 + 2 == 002], [0 * 8 + 3 == 003], [0 * 8 +
        // 4 == 004], ..., [Y * WIDTH + X]
        tmp_array_tile[tmp_thread_block_index_x][tmp_thread_block_diagonal] =
            ptr_array_inputs_received[tmp_thread_global_index_y *
                                          width_received +
                                      tmp_thread_global_index_x];
        // Coalesced Shared Memory:
        // Y[0]: [0][32], [0][33], [0][34], [0][35], [0][36], [0][37], [0][38],
        // ..., [Y][X] = ... Coalesced Global Memory: Y[0]: ... = [0 * 8 + 1 ==
        // 000], [0 * 8 + 1 == 001], [0 * 8 + 2 == 002], [0 * 8 + 3 == 003], [0
        // * 8 + 4 == 004], ..., [Y * WIDTH + X]
        tmp_array_tile[tmp_thread_block_index_x +
                       TILE_WIDTH][tmp_thread_block_diagonal] =
            ptr_array_inputs_received[tmp_thread_global_index_y *
                                          width_received +
                                      tmp_thread_global_index_offSet__x];
        // Coalesced Shared Memory:
        // Y[0 + 32]: [32][0], [32][1], [32][2], [32][3], [32][4], [32][5],
        // [32][6], ..., [Y][X] = ... Coalesced Global Memory: Y[1]: ... = [1 *
        // 8 + 0 == 416], [1 * 384 + 33 == 417], [1 * 384 + 34 == 418], [1 * 384
        // + 35 == 419], [1 * 384 + 36 == 420], ..., [Y * WIDTH + X]
        tmp_array_tile[tmp_thread_block_index_x +
                       2u * TILE_WIDTH][tmp_thread_block_diagonal] =
            ptr_array_inputs_received[tmp_thread_global_index_offSet__y *
                                          width_received +
                                      tmp_thread_global_index_x];
        // Coalesced Shared Memory:
        // Y[0 + 32]: [32][32], [32][33], [32][34], [32][35], [32][36],
        // [32][37], [32][38], ..., [Y][X] = ... Coalesced Global Memory: Y[0 +
        // 32]: ... = [32 * 384 + 32 == 12'320], [32 * 384 + 33 == 12'321], [32
        // * 384 + 34 == 12'322], [32 * 384 + 35 == 12'323], [32 * 384 + 36 ==
        // 12'324], ..., [Y * WIDTH + X]
        tmp_array_tile[tmp_thread_block_index_x +
                       3u * TILE_WIDTH][tmp_thread_block_diagonal] =
            ptr_array_inputs_received[tmp_thread_global_index_offSet__y *
                                          width_received +
                                      tmp_thread_global_index_offSet__x];
      } else if (tmp_thread_global_index_offSet__x < width_received &&
                 tmp_thread_global_index_y < width_received) {
        // Coalesced Shared Memory:
        // Y[0]: [0][0], [0][1], [0][2], [0][3], [0][4], [0][5], [0][6], ...,
        // [Y][X] = ... Coalesced Global Memory: Y[0]: ... = [0 * 8 + 0 == 000],
        // [0 * 8 + 1 == 001], [0 * 8 + 2 == 002], [0 * 8 + 3 == 003], [0 * 8 +
        // 4 == 004], ..., [Y * WIDTH + X]
        tmp_array_tile[tmp_thread_block_index_x][tmp_thread_block_diagonal] =
            ptr_array_inputs_received[tmp_thread_global_index_y *
                                          width_received +
                                      tmp_thread_global_index_x];
        // Coalesced Shared Memory:
        // Y[0]: [0][32], [0][33], [0][34], [0][35], [0][36], [0][37], [0][38],
        // ..., [Y][X] = ... Coalesced Global Memory: Y[0]: ... = [0 * 8 + 0 ==
        // 000], [0 * 8 + 1 == 001], [0 * 8 + 2 == 002], [0 * 8 + 3 == 003], [0
        // * 8 + 4 == 004], ..., [Y * WIDTH + X]
        tmp_array_tile[tmp_thread_block_index_x +
                       TILE_WIDTH][tmp_thread_block_diagonal] =
            ptr_array_inputs_received[tmp_thread_global_index_y *
                                          width_received +
                                      tmp_thread_global_index_offSet__x];
      } else if (tmp_thread_global_index_x < width_received &&
                 tmp_thread_global_index_offSet__y < width_received) {
        // Coalesced Shared Memory:
        // Y[0]: [0][0], [0][1], [0][2], [0][3], [0][4], [0][5], [0][6], ...,
        // [Y][X] = ... Coalesced Global Memory: Y[0]: ... = [0 * 384 + 0 ==
        // 000], [0 * 384 + 1 == 001], [0 * 384 + 2 == 002], [0 * 384 + 3 ==
        // 003], [0 * 384 + 4 == 004], ..., [Y * WIDTH + X]
        tmp_array_tile[tmp_thread_block_index_x][tmp_thread_block_diagonal] =
            ptr_array_inputs_received[tmp_thread_global_index_y *
                                          width_received +
                                      tmp_thread_global_index_x];
        // Coalesced Shared Memory:
        // Y[0 + 32]: [32][0], [32][1], [32][2], [32][3], [32][4], [32][5],
        // [32][6], ..., [Y][X] = ... Coalesced Global Memory: Y[0 + 32]: ... =
        // [32 * 384 + 0 == 12'288], [32 * 384 + 1 == 12'289], [32 * 384 + 2 ==
        // 12'290], [32 * 384 + 3 == 12'291], [32 * 384 + 4 == 12'292], ..., [Y
        // * WIDTH + X]
        tmp_array_tile[tmp_thread_block_index_x +
                       2u * TILE_WIDTH][tmp_thread_block_diagonal] =
            ptr_array_inputs_received[tmp_thread_global_index_offSet__y *
                                          width_received +
                                      tmp_thread_global_index_x];
      } else if (tmp_thread_global_index_x < width_received &&
                 tmp_thread_global_index_y < width_received) {
        // Coalesced Shared Memory:
        // Y[0]: [0][0], [0][1], [0][2], [0][3], [0][4], [0][5], [0][6], ...,
        // [Y][X] = ... Coalesced Global Memory: Y[0]: ... = [0 * 384 + 0 ==
        // 000], [0 * 384 + 1 == 001], [0 * 384 + 2 == 002], [0 * 384 + 3 ==
        // 003], [0 * 384 + 4 == 004], ..., [Y * WIDTH + X]
        tmp_array_tile[tmp_thread_block_index_x][tmp_thread_block_diagonal] =
            ptr_array_inputs_received[tmp_thread_global_index_y *
                                          width_received +
                                      tmp_thread_global_index_x];
      }

      // ThreadBlock synchronization.
      __syncthreads();

      // Transpose X.
      tmp_thread_global_index_x =
          tmp_thread_global_index_block_y + tmp_thread_block_index_x;
      tmp_thread_global_index_offSet__x =
          tmp_thread_global_index_x + TILE_WIDTH;

      // Transpose Y.
      tmp_thread_global_index_y =
          tmp_thread_global_index_block_x + tmp_thread_block_index_y;
      tmp_thread_global_index_offSet__y =
          tmp_thread_global_index_y + TILE_WIDTH;

      if (tmp_thread_global_index_offSet__x < width_received &&
          tmp_thread_global_index_offSet__y < width_received) {
        // Coalesced Shared Memory:
        // Y[0]: ... = [0][0], [1][0], [2][0], [3][0], [4][0], [5][0], [6][0],
        // ..., [X][Y] Coalesced Global Memory: Y[0]: ... = [0 * 384 + 0 ==
        // 000], [0 * 384 + 1 == 001], [0 * 384 + 2 == 002], [0 * 384 + 3 ==
        // 003], [0 * 384 + 4 == 004], ..., [Y * WIDTH + X]
        ptr_array_outputs_received[tmp_thread_global_index_y * width_received +
                                   tmp_thread_global_index_x] =
            tmp_array_tile[tmp_thread_block_index_y][tmp_thread_block_diagonal];
        // Coalesced Shared Memory:
        // Y[0]: [0][32], [0][33], [0][34], [0][35], [0][36], [0][37], [0][38],
        // ..., [Y][X] = ... Coalesced Global Memory: Y[0]: ... = [0 * 384 + 32
        // == 032], [0 * 384 + 33 == 033], [0 * 384 + 34 == 034], [0 * 384 + 35
        // == 035], [0 * 384 + 36 == 036], ..., [Y * WIDTH + X]
        ptr_array_outputs_received[tmp_thread_global_index_y * width_received +
                                   tmp_thread_global_index_offSet__x] =
            tmp_array_tile[tmp_thread_block_index_y + 2u * TILE_WIDTH]
                          [tmp_thread_block_diagonal];
        // Coalesced Shared Memory:
        // Y[0 + 32]: [32][0], [32][1], [32][2], [32][3], [32][4], [32][5],
        // [32][6], ..., [Y][X] = ... Coalesced Global Memory: Y[0 + 32]: ... =
        // [32 * 384 + 0 == 12'288], [32 * 384 + 1 == 12'289], [32 * 384 + 2 ==
        // 12'290], [32 * 384 + 3 == 12'291], [32 * 384 + 4 == 12'292], ..., [Y
        // * WIDTH + X]
        ptr_array_outputs_received[tmp_thread_global_index_offSet__y *
                                       width_received +
                                   tmp_thread_global_index_x] =
            tmp_array_tile[tmp_thread_block_index_y + TILE_WIDTH]
                          [tmp_thread_block_diagonal];
        // Coalesced Shared Memory:
        // Y[0 + 32]: [32][32], [32][33], [32][34], [32][35], [32][36],
        // [32][37], [32][38], ..., [Y][X] = ... Coalesced Global Memory: Y[0 +
        // 32]: ... = [32 * 384 + 32 == 12'320], [32 * 384 + 33 == 12'321], [32
        // * 384 + 34 == 12'322], [32 * 384 + 35 == 12'323], [32 * 384 + 36 ==
        // 12'324], ..., [Y * WIDTH + X]
        ptr_array_outputs_received[tmp_thread_global_index_offSet__y *
                                       width_received +
                                   tmp_thread_global_index_offSet__x] =
            tmp_array_tile[tmp_thread_block_index_y + 3u * TILE_WIDTH]
                          [tmp_thread_block_diagonal];
      } else if (tmp_thread_global_index_offSet__x < width_received &&
                 tmp_thread_global_index_y < width_received) {
        // Coalesced Shared Memory:
        // Y[0]: ... = [0][0], [1][0], [2][0], [3][0], [4][0], [5][0], [6][0],
        // ..., [X][Y] Coalesced Global Memory: Y[0]: ... = [0 * 384 + 0 ==
        // 000], [0 * 384 + 1 == 001], [0 * 384 + 2 == 002], [0 * 384 + 3 ==
        // 003], [0 * 384 + 4 == 004], ..., [Y * WIDTH + X]
        ptr_array_outputs_received[tmp_thread_global_index_y * width_received +
                                   tmp_thread_global_index_x] =
            tmp_array_tile[tmp_thread_block_index_y][tmp_thread_block_diagonal];
        // Coalesced Shared Memory:
        // Y[0]: [0][32], [0][33], [0][34], [0][35], [0][36], [0][37], [0][38],
        // ..., [Y][X] = ... Coalesced Global Memory: Y[0]: ... = [0 * 384 + 32
        // == 032], [0 * 384 + 33 == 033], [0 * 384 + 34 == 034], [0 * 384 + 35
        // == 035], [0 * 384 + 36 == 036], ..., [Y * WIDTH + X]
        ptr_array_outputs_received[tmp_thread_global_index_y * width_received +
                                   tmp_thread_global_index_offSet__x] =
            tmp_array_tile[tmp_thread_block_index_y + 2u * TILE_WIDTH]
                          [tmp_thread_block_diagonal];
      } else if (tmp_thread_global_index_x < width_received &&
                 tmp_thread_global_index_offSet__y < width_received) {
        // Coalesced Shared Memory:
        // Y[0]: ... = [0][0], [1][0], [2][0], [3][0], [4][0], [5][0], [6][0],
        // ..., [X][Y] Coalesced Global Memory: Y[0]: ... = [0 * 384 + 0 ==
        // 000], [0 * 384 + 1 == 001], [0 * 384 + 2 == 002], [0 * 384 + 3 ==
        // 003], [0 * 384 + 4 == 004], ..., [Y * WIDTH + X]
        ptr_array_outputs_received[tmp_thread_global_index_y * width_received +
                                   tmp_thread_global_index_x] =
            tmp_array_tile[tmp_thread_block_index_y][tmp_thread_block_diagonal];
        // Coalesced Shared Memory:
        // Y[0 + 32]: [32][0], [32][1], [32][2], [32][3], [32][4], [32][5],
        // [32][6], ..., [Y][X] = ... Coalesced Global Memory: Y[0 + 32]: ... =
        // [32 * 384 + 0 == 12'288], [32 * 384 + 1 == 12'289], [32 * 384 + 2 ==
        // 12'290], [32 * 384 + 3 == 12'291], [32 * 384 + 4 == 12'292], ..., [Y
        // * WIDTH + X]
        ptr_array_outputs_received[tmp_thread_global_index_offSet__y *
                                       width_received +
                                   tmp_thread_global_index_x] =
            tmp_array_tile[tmp_thread_block_index_y + TILE_WIDTH]
                          [tmp_thread_block_diagonal];
      } else if (tmp_thread_global_index_x < width_received &&
                 tmp_thread_global_index_y < width_received) {
        // Coalesced Shared Memory:
        // Y[0]: ... = [0][0], [1][0], [2][0], [3][0], [4][0], [5][0], [6][0],
        // ..., [X][Y] Coalesced Global Memory: Y[0]: ... = [0 * 384 + 0 ==
        // 000], [0 * 384 + 1 == 001], [0 * 384 + 2 == 002], [0 * 384 + 3 ==
        // 003], [0 * 384 + 4 == 004], ..., [Y * WIDTH + X]
        ptr_array_outputs_received[tmp_thread_global_index_y * width_received +
                                   tmp_thread_global_index_x] =
            tmp_array_tile[tmp_thread_block_index_y][tmp_thread_block_diagonal];
      }

      // ThreadBlock synchronization.
      __syncthreads();

      // Increment X.
      tmp_thread_global_index_block_x += tmp_grid_stride_x;
    }

    // reset X.
    tmp_thread_global_index_block_x = blockIdx.x * TILE_WIDTH * 2u;

    // Increment Y.
    tmp_thread_global_index_block_y += tmp_grid_stride_y;
  }
}

template <typename T>
__device__ void Launch_Transpose_Square(
    size_t size_received, size_t width_received,
    T *const ptr_array_outputs_received,
    T const *const ptr_array_inputs_received,
    struct dim3 const *const ptr_dimension_grid_recieved,
    struct dim3 const *const ptr_dimension_block_recieved) {
  switch (ptr_dimension_block_recieved->x) {
    case 1:
      kernel__Transpose_Square<T, 1u>
          <<<*ptr_dimension_grid_recieved, *ptr_dimension_block_recieved>>>(
              size_received, width_received, ptr_array_outputs_received,
              ptr_array_inputs_received);
      break;
    case 2:
      kernel__Transpose_Square<T, 2u>
          <<<*ptr_dimension_grid_recieved, *ptr_dimension_block_recieved>>>(
              size_received, width_received, ptr_array_outputs_received,
              ptr_array_inputs_received);
      break;
    case 3:
      kernel__Transpose_Square<T, 3u>
          <<<*ptr_dimension_grid_recieved, *ptr_dimension_block_recieved>>>(
              size_received, width_received, ptr_array_outputs_received,
              ptr_array_inputs_received);
      break;
    case 4:
      kernel__Transpose_Square<T, 4u>
          <<<*ptr_dimension_grid_recieved, *ptr_dimension_block_recieved>>>(
              size_received, width_received, ptr_array_outputs_received,
              ptr_array_inputs_received);
      break;
    case 5:
      kernel__Transpose_Square<T, 5u>
          <<<*ptr_dimension_grid_recieved, *ptr_dimension_block_recieved>>>(
              size_received, width_received, ptr_array_outputs_received,
              ptr_array_inputs_received);
      break;
    case 6:
      kernel__Transpose_Square<T, 6u>
          <<<*ptr_dimension_grid_recieved, *ptr_dimension_block_recieved>>>(
              size_received, width_received, ptr_array_outputs_received,
              ptr_array_inputs_received);
      break;
    case 7:
      kernel__Transpose_Square<T, 7u>
          <<<*ptr_dimension_grid_recieved, *ptr_dimension_block_recieved>>>(
              size_received, width_received, ptr_array_outputs_received,
              ptr_array_inputs_received);
      break;
    case 8:
      kernel__Transpose_Square<T, 8u>
          <<<*ptr_dimension_grid_recieved, *ptr_dimension_block_recieved>>>(
              size_received, width_received, ptr_array_outputs_received,
              ptr_array_inputs_received);
      break;
    case 9:
      kernel__Transpose_Square<T, 9u>
          <<<*ptr_dimension_grid_recieved, *ptr_dimension_block_recieved>>>(
              size_received, width_received, ptr_array_outputs_received,
              ptr_array_inputs_received);
      break;
    case 10:
      kernel__Transpose_Square<T, 10u>
          <<<*ptr_dimension_grid_recieved, *ptr_dimension_block_recieved>>>(
              size_received, width_received, ptr_array_outputs_received,
              ptr_array_inputs_received);
      break;
    case 11:
      kernel__Transpose_Square<T, 11u>
          <<<*ptr_dimension_grid_recieved, *ptr_dimension_block_recieved>>>(
              size_received, width_received, ptr_array_outputs_received,
              ptr_array_inputs_received);
      break;
    case 12:
      kernel__Transpose_Square<T, 12u>
          <<<*ptr_dimension_grid_recieved, *ptr_dimension_block_recieved>>>(
              size_received, width_received, ptr_array_outputs_received,
              ptr_array_inputs_received);
      break;
    case 13:
      kernel__Transpose_Square<T, 13u>
          <<<*ptr_dimension_grid_recieved, *ptr_dimension_block_recieved>>>(
              size_received, width_received, ptr_array_outputs_received,
              ptr_array_inputs_received);
      break;
    case 14:
      kernel__Transpose_Square<T, 14u>
          <<<*ptr_dimension_grid_recieved, *ptr_dimension_block_recieved>>>(
              size_received, width_received, ptr_array_outputs_received,
              ptr_array_inputs_received);
      break;
    case 15:
      kernel__Transpose_Square<T, 15u>
          <<<*ptr_dimension_grid_recieved, *ptr_dimension_block_recieved>>>(
              size_received, width_received, ptr_array_outputs_received,
              ptr_array_inputs_received);
      break;
    case 16:
      kernel__Transpose_Square<T, 16u>
          <<<*ptr_dimension_grid_recieved, *ptr_dimension_block_recieved>>>(
              size_received, width_received, ptr_array_outputs_received,
              ptr_array_inputs_received);
      break;
    case 17:
      kernel__Transpose_Square<T, 17u>
          <<<*ptr_dimension_grid_recieved, *ptr_dimension_block_recieved>>>(
              size_received, width_received, ptr_array_outputs_received,
              ptr_array_inputs_received);
      break;
    case 18:
      kernel__Transpose_Square<T, 18u>
          <<<*ptr_dimension_grid_recieved, *ptr_dimension_block_recieved>>>(
              size_received, width_received, ptr_array_outputs_received,
              ptr_array_inputs_received);
      break;
    case 19:
      kernel__Transpose_Square<T, 19u>
          <<<*ptr_dimension_grid_recieved, *ptr_dimension_block_recieved>>>(
              size_received, width_received, ptr_array_outputs_received,
              ptr_array_inputs_received);
      break;
    case 20:
      kernel__Transpose_Square<T, 20u>
          <<<*ptr_dimension_grid_recieved, *ptr_dimension_block_recieved>>>(
              size_received, width_received, ptr_array_outputs_received,
              ptr_array_inputs_received);
      break;
    case 21:
      kernel__Transpose_Square<T, 21u>
          <<<*ptr_dimension_grid_recieved, *ptr_dimension_block_recieved>>>(
              size_received, width_received, ptr_array_outputs_received,
              ptr_array_inputs_received);
      break;
    case 22:
      kernel__Transpose_Square<T, 22u>
          <<<*ptr_dimension_grid_recieved, *ptr_dimension_block_recieved>>>(
              size_received, width_received, ptr_array_outputs_received,
              ptr_array_inputs_received);
      break;
    case 23:
      kernel__Transpose_Square<T, 23u>
          <<<*ptr_dimension_grid_recieved, *ptr_dimension_block_recieved>>>(
              size_received, width_received, ptr_array_outputs_received,
              ptr_array_inputs_received);
      break;
    case 24:
      kernel__Transpose_Square<T, 24u>
          <<<*ptr_dimension_grid_recieved, *ptr_dimension_block_recieved>>>(
              size_received, width_received, ptr_array_outputs_received,
              ptr_array_inputs_received);
      break;
    case 25:
      kernel__Transpose_Square<T, 25u>
          <<<*ptr_dimension_grid_recieved, *ptr_dimension_block_recieved>>>(
              size_received, width_received, ptr_array_outputs_received,
              ptr_array_inputs_received);
      break;
    case 26:
      kernel__Transpose_Square<T, 26u>
          <<<*ptr_dimension_grid_recieved, *ptr_dimension_block_recieved>>>(
              size_received, width_received, ptr_array_outputs_received,
              ptr_array_inputs_received);
      break;
    case 27:
      kernel__Transpose_Square<T, 27u>
          <<<*ptr_dimension_grid_recieved, *ptr_dimension_block_recieved>>>(
              size_received, width_received, ptr_array_outputs_received,
              ptr_array_inputs_received);
      break;
    case 28:
      kernel__Transpose_Square<T, 28u>
          <<<*ptr_dimension_grid_recieved, *ptr_dimension_block_recieved>>>(
              size_received, width_received, ptr_array_outputs_received,
              ptr_array_inputs_received);
      break;
    case 29:
      kernel__Transpose_Square<T, 29u>
          <<<*ptr_dimension_grid_recieved, *ptr_dimension_block_recieved>>>(
              size_received, width_received, ptr_array_outputs_received,
              ptr_array_inputs_received);
      break;
    case 30:
      kernel__Transpose_Square<T, 30u>
          <<<*ptr_dimension_grid_recieved, *ptr_dimension_block_recieved>>>(
              size_received, width_received, ptr_array_outputs_received,
              ptr_array_inputs_received);
      break;
    case 31:
      kernel__Transpose_Square<T, 31u>
          <<<*ptr_dimension_grid_recieved, *ptr_dimension_block_recieved>>>(
              size_received, width_received, ptr_array_outputs_received,
              ptr_array_inputs_received);
      break;
    case 32:
      kernel__Transpose_Square<T, 32u>
          <<<*ptr_dimension_grid_recieved, *ptr_dimension_block_recieved>>>(
              size_received, width_received, ptr_array_outputs_received,
              ptr_array_inputs_received);
      break;
    default:
      ERR(L"Invalid dimension %ux%u!",
                   ptr_dimension_block_recieved->x,
                   ptr_dimension_block_recieved->y);
      break;
  }
}

template <typename T>
__device__ void Transpose_Square(
    size_t const size_received, size_t const width_received,
    T *const ptr_array_outputs_received,
    T const *const ptr_array_inputs_received,
    struct dim3 const *const ptr_dimension_grid_recieved,
    struct dim3 const *const ptr_dimension_block_recieved) {
  /* E.g:
      Width = 2;

      Output:
          Index = row * width + column;
          0 * 2 + 0 = [0]; 0 * 2 + 1 = [1];
          1 * 2 + 0 = [2]; 1 * 2 + 1 = [3];

      Input:
          Index = column * width + row;
          0 * 2 + 0 = [0]; 1 * 2 + 0 = [2];
          0 * 2 + 1 = [1]; 1 * 2 + 1 = [3]; */

  if (USE_PARALLEL && size_received >= warpSize) {
    Launch_Transpose_Square<T>(
        size_received, width_received, ptr_array_outputs_received,
        ptr_array_inputs_received, ptr_dimension_grid_recieved,
        ptr_dimension_block_recieved);
  } else {
    for (size_t row(0u), column(0u); column != width_received; ++column) {
      for (row = 0u; row != width_received; ++row) {
        ptr_array_outputs_received[row * width_received + column] =
            ptr_array_inputs_received[column * width_received + row];
      }
    }

    /*
    INFO(L"Input %ux%u: " , width_received, width_received);
    for(size_t column(0u),
                           row(0u); row != width_received; ++row)
    {
        for(column = 0u; column != width_received; ++column)
        {
            INFO(L"[%f] ", ptr_array_inputs_received[column *
    width_received + row]);
        }

        INFO(L"");
    }

    INFO(L"Output %ux%u: " , width_received, width_received);
    for(size_t column(0u),
                           row(0u); row != width_received; ++row)
    {
        for(column = 0u; column != width_received; ++column)
        {
            INFO(L"[%f] ", ptr_array_outputs_received[column *
    width_received + row]);
        }

        INFO(L"");
    }
    */
  }
}

// TODO: Fix. Some dimension like 25x25 have bank conflict.
template <typename T, size_t TILE_WIDTH>
__global__ void kernel__Transpose_Rectangular(
    size_t const size_received, size_t const rows,
    size_t const cols, T *const ptr_array_outputs_received,
    T const *const ptr_array_inputs_received) {
  size_t const &tmp_thread_block_index_x(threadIdx.x),
      &tmp_thread_block_index_y(threadIdx.y),
      tmp_thread_block_diagonal(
          (tmp_thread_block_index_x + tmp_thread_block_index_y) % TILE_WIDTH),
      tmp_grid_stride_x(gridDim.x * TILE_WIDTH * 2u),
      tmp_grid_stride_y(gridDim.y * TILE_WIDTH * 2u);
  size_t tmp_thread_global_index_block_x(blockIdx.x * TILE_WIDTH * 2u),
      tmp_thread_global_index_block_y(blockIdx.y * TILE_WIDTH * 2u),
      tmp_thread_global_index_x, tmp_thread_global_index_y,
      tmp_thread_global_index_offSet__x, tmp_thread_global_index_offSet__y;

  __shared__ T tmp_array_tile[TILE_WIDTH * 4u][TILE_WIDTH];

  while (tmp_thread_global_index_block_y < cols) {
    while (tmp_thread_global_index_block_x < rows) {
      // Coalesced index X.
      // 0 * 32 * 2 + [0...1...31] = 0 + [0...1...31]
      // 1 * 32 * 2 + [0...1...31] = 64 + [0...1...31]
      tmp_thread_global_index_x =
          tmp_thread_global_index_block_x + tmp_thread_block_index_x;
      tmp_thread_global_index_offSet__x =
          tmp_thread_global_index_x + TILE_WIDTH;

      // Coalesced index Y.
      // 0 * 32 * 2 + [0...1...31] = 0 + [0...1...31]
      // 1 * 32 * 2 + [0...1...31] = 64 + [0...1...31]
      tmp_thread_global_index_y =
          tmp_thread_global_index_block_y + tmp_thread_block_index_y;
      tmp_thread_global_index_offSet__y =
          tmp_thread_global_index_y + TILE_WIDTH;

      if (tmp_thread_global_index_offSet__x < rows &&
          tmp_thread_global_index_offSet__y < cols) {
        // Coalesced Shared Memory:
        // Y[0]: [0][0], [0][1], [0][2], [0][3], [0][4], [0][5], [0][6], ...,
        // [Y][X] = ... Coalesced Global Memory: Y[0]: ... = [0 * 8 + 0 == 000],
        // [0 * 8 + 1 == 001], [0 * 8 + 2 == 002], [0 * 8 + 3 == 003], [0 * 8 +
        // 4 == 004], ..., [Y * WIDTH + X]
        tmp_array_tile[tmp_thread_block_index_x][tmp_thread_block_diagonal] =
            ptr_array_inputs_received[tmp_thread_global_index_y *
                                          rows +
                                      tmp_thread_global_index_x];
        // Coalesced Shared Memory:
        // Y[0]: [0][32], [0][33], [0][34], [0][35], [0][36], [0][37], [0][38],
        // ..., [Y][X] = ... Coalesced Global Memory: Y[0]: ... = [0 * 8 + 1 ==
        // 000], [0 * 8 + 1 == 001], [0 * 8 + 2 == 002], [0 * 8 + 3 == 003], [0
        // * 8 + 4 == 004], ..., [Y * WIDTH + X]
        tmp_array_tile[tmp_thread_block_index_x +
                       TILE_WIDTH][tmp_thread_block_diagonal] =
            ptr_array_inputs_received[tmp_thread_global_index_y *
                                          rows +
                                      tmp_thread_global_index_offSet__x];
        // Coalesced Shared Memory:
        // Y[0 + 32]: [32][0], [32][1], [32][2], [32][3], [32][4], [32][5],
        // [32][6], ..., [Y][X] = ... Coalesced Global Memory: Y[1]: ... = [1 *
        // 8 + 0 == 416], [1 * 384 + 33 == 417], [1 * 384 + 34 == 418], [1 * 384
        // + 35 == 419], [1 * 384 + 36 == 420], ..., [Y * WIDTH + X]
        tmp_array_tile[tmp_thread_block_index_x +
                       2u * TILE_WIDTH][tmp_thread_block_diagonal] =
            ptr_array_inputs_received[tmp_thread_global_index_offSet__y *
                                          rows +
                                      tmp_thread_global_index_x];
        // Coalesced Shared Memory:
        // Y[0 + 32]: [32][32], [32][33], [32][34], [32][35], [32][36],
        // [32][37], [32][38], ..., [Y][X] = ... Coalesced Global Memory: Y[0 +
        // 32]: ... = [32 * 384 + 32 == 12'320], [32 * 384 + 33 == 12'321], [32
        // * 384 + 34 == 12'322], [32 * 384 + 35 == 12'323], [32 * 384 + 36 ==
        // 12'324], ..., [Y * WIDTH + X]
        tmp_array_tile[tmp_thread_block_index_x +
                       3u * TILE_WIDTH][tmp_thread_block_diagonal] =
            ptr_array_inputs_received[tmp_thread_global_index_offSet__y *
                                          rows +
                                      tmp_thread_global_index_offSet__x];
      } else if (tmp_thread_global_index_offSet__x < rows &&
                 tmp_thread_global_index_y < cols) {
        // Coalesced Shared Memory:
        // Y[0]: [0][0], [0][1], [0][2], [0][3], [0][4], [0][5], [0][6], ...,
        // [Y][X] = ... Coalesced Global Memory: Y[0]: ... = [0 * 8 + 0 == 000],
        // [0 * 8 + 1 == 001], [0 * 8 + 2 == 002], [0 * 8 + 3 == 003], [0 * 8 +
        // 4 == 004], ..., [Y * WIDTH + X]
        tmp_array_tile[tmp_thread_block_index_x][tmp_thread_block_diagonal] =
            ptr_array_inputs_received[tmp_thread_global_index_y *
                                          rows +
                                      tmp_thread_global_index_x];
        // Coalesced Shared Memory:
        // Y[0]: [0][32], [0][33], [0][34], [0][35], [0][36], [0][37], [0][38],
        // ..., [Y][X] = ... Coalesced Global Memory: Y[0]: ... = [0 * 8 + 0 ==
        // 000], [0 * 8 + 1 == 001], [0 * 8 + 2 == 002], [0 * 8 + 3 == 003], [0
        // * 8 + 4 == 004], ..., [Y * WIDTH + X]
        tmp_array_tile[tmp_thread_block_index_x +
                       TILE_WIDTH][tmp_thread_block_diagonal] =
            ptr_array_inputs_received[tmp_thread_global_index_y *
                                          rows +
                                      tmp_thread_global_index_offSet__x];
      } else if (tmp_thread_global_index_x < rows &&
                 tmp_thread_global_index_offSet__y < cols) {
        // Coalesced Shared Memory:
        // Y[0]: [0][0], [0][1], [0][2], [0][3], [0][4], [0][5], [0][6], ...,
        // [Y][X] = ... Coalesced Global Memory: Y[0]: ... = [0 * 384 + 0 ==
        // 000], [0 * 384 + 1 == 001], [0 * 384 + 2 == 002], [0 * 384 + 3 ==
        // 003], [0 * 384 + 4 == 004], ..., [Y * WIDTH + X]
        tmp_array_tile[tmp_thread_block_index_x][tmp_thread_block_diagonal] =
            ptr_array_inputs_received[tmp_thread_global_index_y *
                                          rows +
                                      tmp_thread_global_index_x];
        // Coalesced Shared Memory:
        // Y[0 + 32]: [32][0], [32][1], [32][2], [32][3], [32][4], [32][5],
        // [32][6], ..., [Y][X] = ... Coalesced Global Memory: Y[0 + 32]: ... =
        // [32 * 384 + 0 == 12'288], [32 * 384 + 1 == 12'289], [32 * 384 + 2 ==
        // 12'290], [32 * 384 + 3 == 12'291], [32 * 384 + 4 == 12'292], ..., [Y
        // * WIDTH + X]
        tmp_array_tile[tmp_thread_block_index_x +
                       2u * TILE_WIDTH][tmp_thread_block_diagonal] =
            ptr_array_inputs_received[tmp_thread_global_index_offSet__y *
                                          rows +
                                      tmp_thread_global_index_x];
      } else if (tmp_thread_global_index_x < rows &&
                 tmp_thread_global_index_y < cols) {
        // Coalesced Shared Memory:
        // Y[0]: [0][0], [0][1], [0][2], [0][3], [0][4], [0][5], [0][6], ...,
        // [Y][X] = ... Coalesced Global Memory: Y[0]: ... = [0 * 384 + 0 ==
        // 000], [0 * 384 + 1 == 001], [0 * 384 + 2 == 002], [0 * 384 + 3 ==
        // 003], [0 * 384 + 4 == 004], ..., [Y * WIDTH + X]
        tmp_array_tile[tmp_thread_block_index_x][tmp_thread_block_diagonal] =
            ptr_array_inputs_received[tmp_thread_global_index_y *
                                          rows +
                                      tmp_thread_global_index_x];
      }

      // ThreadBlock synchronization.
      __syncthreads();

      // Transpose X.
      tmp_thread_global_index_x =
          tmp_thread_global_index_block_y + tmp_thread_block_index_x;
      tmp_thread_global_index_offSet__x =
          tmp_thread_global_index_x + TILE_WIDTH;

      // Transpose Y.
      tmp_thread_global_index_y =
          tmp_thread_global_index_block_x + tmp_thread_block_index_y;
      tmp_thread_global_index_offSet__y =
          tmp_thread_global_index_y + TILE_WIDTH;

      if (tmp_thread_global_index_offSet__x < cols &&
          tmp_thread_global_index_offSet__y < rows) {
        // Coalesced Shared Memory:
        // Y[0]: ... = [0][0], [1][0], [2][0], [3][0], [4][0], [5][0], [6][0],
        // ..., [X][Y] Coalesced Global Memory: Y[0]: ... = [0 * 384 + 0 ==
        // 000], [0 * 384 + 1 == 001], [0 * 384 + 2 == 002], [0 * 384 + 3 ==
        // 003], [0 * 384 + 4 == 004], ..., [Y * WIDTH + X]
        ptr_array_outputs_received[tmp_thread_global_index_y *
                                       cols +
                                   tmp_thread_global_index_x] =
            tmp_array_tile[tmp_thread_block_index_y][tmp_thread_block_diagonal];
        // Coalesced Shared Memory:
        // Y[0]: [0][32], [0][33], [0][34], [0][35], [0][36], [0][37], [0][38],
        // ..., [Y][X] = ... Coalesced Global Memory: Y[0]: ... = [0 * 384 + 32
        // == 032], [0 * 384 + 33 == 033], [0 * 384 + 34 == 034], [0 * 384 + 35
        // == 035], [0 * 384 + 36 == 036], ..., [Y * WIDTH + X]
        ptr_array_outputs_received[tmp_thread_global_index_y *
                                       cols +
                                   tmp_thread_global_index_offSet__x] =
            tmp_array_tile[tmp_thread_block_index_y + 2u * TILE_WIDTH]
                          [tmp_thread_block_diagonal];
        // Coalesced Shared Memory:
        // Y[0 + 32]: [32][0], [32][1], [32][2], [32][3], [32][4], [32][5],
        // [32][6], ..., [Y][X] = ... Coalesced Global Memory: Y[0 + 32]: ... =
        // [32 * 384 + 0 == 12'288], [32 * 384 + 1 == 12'289], [32 * 384 + 2 ==
        // 12'290], [32 * 384 + 3 == 12'291], [32 * 384 + 4 == 12'292], ..., [Y
        // * WIDTH + X]
        ptr_array_outputs_received[tmp_thread_global_index_offSet__y *
                                       cols +
                                   tmp_thread_global_index_x] =
            tmp_array_tile[tmp_thread_block_index_y + TILE_WIDTH]
                          [tmp_thread_block_diagonal];
        // Coalesced Shared Memory:
        // Y[0 + 32]: [32][32], [32][33], [32][34], [32][35], [32][36],
        // [32][37], [32][38], ..., [Y][X] = ... Coalesced Global Memory: Y[0 +
        // 32]: ... = [32 * 384 + 32 == 12'320], [32 * 384 + 33 == 12'321], [32
        // * 384 + 34 == 12'322], [32 * 384 + 35 == 12'323], [32 * 384 + 36 ==
        // 12'324], ..., [Y * WIDTH + X]
        ptr_array_outputs_received[tmp_thread_global_index_offSet__y *
                                       cols +
                                   tmp_thread_global_index_offSet__x] =
            tmp_array_tile[tmp_thread_block_index_y + 3u * TILE_WIDTH]
                          [tmp_thread_block_diagonal];
      } else if (tmp_thread_global_index_offSet__x < cols &&
                 tmp_thread_global_index_y < rows) {
        // Coalesced Shared Memory:
        // Y[0]: ... = [0][0], [1][0], [2][0], [3][0], [4][0], [5][0], [6][0],
        // ..., [X][Y] Coalesced Global Memory: Y[0]: ... = [0 * 384 + 0 ==
        // 000], [0 * 384 + 1 == 001], [0 * 384 + 2 == 002], [0 * 384 + 3 ==
        // 003], [0 * 384 + 4 == 004], ..., [Y * WIDTH + X]
        ptr_array_outputs_received[tmp_thread_global_index_y *
                                       cols +
                                   tmp_thread_global_index_x] =
            tmp_array_tile[tmp_thread_block_index_y][tmp_thread_block_diagonal];
        // Coalesced Shared Memory:
        // Y[0]: [0][32], [0][33], [0][34], [0][35], [0][36], [0][37], [0][38],
        // ..., [Y][X] = ... Coalesced Global Memory: Y[0]: ... = [0 * 384 + 32
        // == 032], [0 * 384 + 33 == 033], [0 * 384 + 34 == 034], [0 * 384 + 35
        // == 035], [0 * 384 + 36 == 036], ..., [Y * WIDTH + X]
        ptr_array_outputs_received[tmp_thread_global_index_y *
                                       cols +
                                   tmp_thread_global_index_offSet__x] =
            tmp_array_tile[tmp_thread_block_index_y + 2u * TILE_WIDTH]
                          [tmp_thread_block_diagonal];
      } else if (tmp_thread_global_index_x < cols &&
                 tmp_thread_global_index_offSet__y < rows) {
        // Coalesced Shared Memory:
        // Y[0]: ... = [0][0], [1][0], [2][0], [3][0], [4][0], [5][0], [6][0],
        // ..., [X][Y] Coalesced Global Memory: Y[0]: ... = [0 * 384 + 0 ==
        // 000], [0 * 384 + 1 == 001], [0 * 384 + 2 == 002], [0 * 384 + 3 ==
        // 003], [0 * 384 + 4 == 004], ..., [Y * WIDTH + X]
        ptr_array_outputs_received[tmp_thread_global_index_y *
                                       cols +
                                   tmp_thread_global_index_x] =
            tmp_array_tile[tmp_thread_block_index_y][tmp_thread_block_diagonal];
        // Coalesced Shared Memory:
        // Y[0 + 32]: [32][0], [32][1], [32][2], [32][3], [32][4], [32][5],
        // [32][6], ..., [Y][X] = ... Coalesced Global Memory: Y[0 + 32]: ... =
        // [32 * 384 + 0 == 12'288], [32 * 384 + 1 == 12'289], [32 * 384 + 2 ==
        // 12'290], [32 * 384 + 3 == 12'291], [32 * 384 + 4 == 12'292], ..., [Y
        // * WIDTH + X]
        ptr_array_outputs_received[tmp_thread_global_index_offSet__y *
                                       cols +
                                   tmp_thread_global_index_x] =
            tmp_array_tile[tmp_thread_block_index_y + TILE_WIDTH]
                          [tmp_thread_block_diagonal];
      } else if (tmp_thread_global_index_x < cols &&
                 tmp_thread_global_index_y < rows) {
        // Coalesced Shared Memory:
        // Y[0]: ... = [0][0], [1][0], [2][0], [3][0], [4][0], [5][0], [6][0],
        // ..., [X][Y] Coalesced Global Memory: Y[0]: ... = [0 * 384 + 0 ==
        // 000], [0 * 384 + 1 == 001], [0 * 384 + 2 == 002], [0 * 384 + 3 ==
        // 003], [0 * 384 + 4 == 004], ..., [Y * WIDTH + X]
        ptr_array_outputs_received[tmp_thread_global_index_y *
                                       cols +
                                   tmp_thread_global_index_x] =
            tmp_array_tile[tmp_thread_block_index_y][tmp_thread_block_diagonal];
      }

      // ThreadBlock synchronization.
      __syncthreads();

      // Increment X.
      tmp_thread_global_index_block_x += tmp_grid_stride_x;
    }

    // reset X.
    tmp_thread_global_index_block_x = blockIdx.x * TILE_WIDTH * 2u;

    // Increment Y.
    tmp_thread_global_index_block_y += tmp_grid_stride_y;
  }
}

template <typename T>
__device__ void Launch_Transpose_Rectangular(
    size_t size_received, size_t const rows,
    size_t const cols, T *const ptr_array_outputs_received,
    T const *const ptr_array_inputs_received,
    struct dim3 const *const ptr_dimension_grid_recieved,
    struct dim3 const *const ptr_dimension_block_recieved) {
  switch (ptr_dimension_block_recieved->x) {
    case 1:
      kernel__Transpose_Rectangular<T, 1u>
          <<<*ptr_dimension_grid_recieved, *ptr_dimension_block_recieved>>>(
              size_received, rows, cols,
              ptr_array_outputs_received, ptr_array_inputs_received);
      break;
    case 2:
      kernel__Transpose_Rectangular<T, 2u>
          <<<*ptr_dimension_grid_recieved, *ptr_dimension_block_recieved>>>(
              size_received, rows, cols,
              ptr_array_outputs_received, ptr_array_inputs_received);
      break;
    case 3:
      kernel__Transpose_Rectangular<T, 3u>
          <<<*ptr_dimension_grid_recieved, *ptr_dimension_block_recieved>>>(
              size_received, rows, cols,
              ptr_array_outputs_received, ptr_array_inputs_received);
      break;
    case 4:
      kernel__Transpose_Rectangular<T, 4u>
          <<<*ptr_dimension_grid_recieved, *ptr_dimension_block_recieved>>>(
              size_received, rows, cols,
              ptr_array_outputs_received, ptr_array_inputs_received);
      break;
    case 5:
      kernel__Transpose_Rectangular<T, 5u>
          <<<*ptr_dimension_grid_recieved, *ptr_dimension_block_recieved>>>(
              size_received, rows, cols,
              ptr_array_outputs_received, ptr_array_inputs_received);
      break;
    case 6:
      kernel__Transpose_Rectangular<T, 6u>
          <<<*ptr_dimension_grid_recieved, *ptr_dimension_block_recieved>>>(
              size_received, rows, cols,
              ptr_array_outputs_received, ptr_array_inputs_received);
      break;
    case 7:
      kernel__Transpose_Rectangular<T, 7u>
          <<<*ptr_dimension_grid_recieved, *ptr_dimension_block_recieved>>>(
              size_received, rows, cols,
              ptr_array_outputs_received, ptr_array_inputs_received);
      break;
    case 8:
      kernel__Transpose_Rectangular<T, 8u>
          <<<*ptr_dimension_grid_recieved, *ptr_dimension_block_recieved>>>(
              size_received, rows, cols,
              ptr_array_outputs_received, ptr_array_inputs_received);
      break;
    case 9:
      kernel__Transpose_Rectangular<T, 9u>
          <<<*ptr_dimension_grid_recieved, *ptr_dimension_block_recieved>>>(
              size_received, rows, cols,
              ptr_array_outputs_received, ptr_array_inputs_received);
      break;
    case 10:
      kernel__Transpose_Rectangular<T, 10u>
          <<<*ptr_dimension_grid_recieved, *ptr_dimension_block_recieved>>>(
              size_received, rows, cols,
              ptr_array_outputs_received, ptr_array_inputs_received);
      break;
    case 11:
      kernel__Transpose_Rectangular<T, 11u>
          <<<*ptr_dimension_grid_recieved, *ptr_dimension_block_recieved>>>(
              size_received, rows, cols,
              ptr_array_outputs_received, ptr_array_inputs_received);
      break;
    case 12:
      kernel__Transpose_Rectangular<T, 12u>
          <<<*ptr_dimension_grid_recieved, *ptr_dimension_block_recieved>>>(
              size_received, rows, cols,
              ptr_array_outputs_received, ptr_array_inputs_received);
      break;
    case 13:
      kernel__Transpose_Rectangular<T, 13u>
          <<<*ptr_dimension_grid_recieved, *ptr_dimension_block_recieved>>>(
              size_received, rows, cols,
              ptr_array_outputs_received, ptr_array_inputs_received);
      break;
    case 14:
      kernel__Transpose_Rectangular<T, 14u>
          <<<*ptr_dimension_grid_recieved, *ptr_dimension_block_recieved>>>(
              size_received, rows, cols,
              ptr_array_outputs_received, ptr_array_inputs_received);
      break;
    case 15:
      kernel__Transpose_Rectangular<T, 15u>
          <<<*ptr_dimension_grid_recieved, *ptr_dimension_block_recieved>>>(
              size_received, rows, cols,
              ptr_array_outputs_received, ptr_array_inputs_received);
      break;
    case 16:
      kernel__Transpose_Rectangular<T, 16u>
          <<<*ptr_dimension_grid_recieved, *ptr_dimension_block_recieved>>>(
              size_received, rows, cols,
              ptr_array_outputs_received, ptr_array_inputs_received);
      break;
    case 17:
      kernel__Transpose_Rectangular<T, 17u>
          <<<*ptr_dimension_grid_recieved, *ptr_dimension_block_recieved>>>(
              size_received, rows, cols,
              ptr_array_outputs_received, ptr_array_inputs_received);
      break;
    case 18:
      kernel__Transpose_Rectangular<T, 18u>
          <<<*ptr_dimension_grid_recieved, *ptr_dimension_block_recieved>>>(
              size_received, rows, cols,
              ptr_array_outputs_received, ptr_array_inputs_received);
      break;
    case 19:
      kernel__Transpose_Rectangular<T, 19u>
          <<<*ptr_dimension_grid_recieved, *ptr_dimension_block_recieved>>>(
              size_received, rows, cols,
              ptr_array_outputs_received, ptr_array_inputs_received);
      break;
    case 20:
      kernel__Transpose_Rectangular<T, 20u>
          <<<*ptr_dimension_grid_recieved, *ptr_dimension_block_recieved>>>(
              size_received, rows, cols,
              ptr_array_outputs_received, ptr_array_inputs_received);
      break;
    case 21:
      kernel__Transpose_Rectangular<T, 21u>
          <<<*ptr_dimension_grid_recieved, *ptr_dimension_block_recieved>>>(
              size_received, rows, cols,
              ptr_array_outputs_received, ptr_array_inputs_received);
      break;
    case 22:
      kernel__Transpose_Rectangular<T, 22u>
          <<<*ptr_dimension_grid_recieved, *ptr_dimension_block_recieved>>>(
              size_received, rows, cols,
              ptr_array_outputs_received, ptr_array_inputs_received);
      break;
    case 23:
      kernel__Transpose_Rectangular<T, 23u>
          <<<*ptr_dimension_grid_recieved, *ptr_dimension_block_recieved>>>(
              size_received, rows, cols,
              ptr_array_outputs_received, ptr_array_inputs_received);
      break;
    case 24:
      kernel__Transpose_Rectangular<T, 24u>
          <<<*ptr_dimension_grid_recieved, *ptr_dimension_block_recieved>>>(
              size_received, rows, cols,
              ptr_array_outputs_received, ptr_array_inputs_received);
      break;
    case 25:
      kernel__Transpose_Rectangular<T, 25u>
          <<<*ptr_dimension_grid_recieved, *ptr_dimension_block_recieved>>>(
              size_received, rows, cols,
              ptr_array_outputs_received, ptr_array_inputs_received);
      break;
    case 26:
      kernel__Transpose_Rectangular<T, 26u>
          <<<*ptr_dimension_grid_recieved, *ptr_dimension_block_recieved>>>(
              size_received, rows, cols,
              ptr_array_outputs_received, ptr_array_inputs_received);
      break;
    case 27:
      kernel__Transpose_Rectangular<T, 27u>
          <<<*ptr_dimension_grid_recieved, *ptr_dimension_block_recieved>>>(
              size_received, rows, cols,
              ptr_array_outputs_received, ptr_array_inputs_received);
      break;
    case 28:
      kernel__Transpose_Rectangular<T, 28u>
          <<<*ptr_dimension_grid_recieved, *ptr_dimension_block_recieved>>>(
              size_received, rows, cols,
              ptr_array_outputs_received, ptr_array_inputs_received);
      break;
    case 29:
      kernel__Transpose_Rectangular<T, 29u>
          <<<*ptr_dimension_grid_recieved, *ptr_dimension_block_recieved>>>(
              size_received, rows, cols,
              ptr_array_outputs_received, ptr_array_inputs_received);
      break;
    case 30:
      kernel__Transpose_Rectangular<T, 30u>
          <<<*ptr_dimension_grid_recieved, *ptr_dimension_block_recieved>>>(
              size_received, rows, cols,
              ptr_array_outputs_received, ptr_array_inputs_received);
      break;
    case 31:
      kernel__Transpose_Rectangular<T, 31u>
          <<<*ptr_dimension_grid_recieved, *ptr_dimension_block_recieved>>>(
              size_received, rows, cols,
              ptr_array_outputs_received, ptr_array_inputs_received);
      break;
    case 32:
      kernel__Transpose_Rectangular<T, 32u>
          <<<*ptr_dimension_grid_recieved, *ptr_dimension_block_recieved>>>(
              size_received, rows, cols,
              ptr_array_outputs_received, ptr_array_inputs_received);
      break;
    default:
      ERR(L"Invalid dimension %ux%u!",
                   ptr_dimension_block_recieved->x,
                   ptr_dimension_block_recieved->y);
      break;
  }
}

template <typename T>
__device__ void Transpose_Rectangular(
    size_t const size_received, size_t const rows_length_received,
    size_t const columns_length_received, T *const ptr_array_outputs_received,
    T const *const ptr_array_inputs_received,
    struct dim3 const *const ptr_dimension_grid_recieved,
    struct dim3 const *const ptr_dimension_block_recieved) {
  /* E.g:
      columns_length = 3;
      rows_length = 2;

      Output:
          Index = row * columns_length + column;
          0 * 3 + 0 = [0]; 0 * 3 + 1 = [1]; 0 * 3 + 2= [2];
          1 * 3 + 0 = [3]; 1 * 3 + 1 = [4]; 1 * 3 + 2 = [5];

      Input:
          Index = column * rows_length + row;
          0 * 2 + 0 = [0]; 1 * 2 + 0 = [2]; 2 * 2 + 0 = [4];
          0 * 2 + 1 = [1]; 1 * 2 + 1 = [3]; 2 * 2 + 1 = [5]; */

  if (USE_PARALLEL && size_received >= warpSize) {
    Launch_Transpose_Rectangular<T>(
        size_received, rows_length_received, columns_length_received,
        ptr_array_outputs_received, ptr_array_inputs_received,
        ptr_dimension_grid_recieved, ptr_dimension_block_recieved);
  } else {
    for (size_t row(0u), column(0u); column != columns_length_received;
         ++column) {
      for (row = 0u; row != rows_length_received; ++row) {
        ptr_array_outputs_received[row * columns_length_received + column] =
            ptr_array_inputs_received[column * rows_length_received + row];
      }
    }

    /*
    INFO(L"Input %ux%u: " , rows_length_received,
    columns_length_received); for(size_t column(0u), row(0u); row !=
    rows_length_received; ++row)
    {
        for(column = 0u; column != columns_length_received; ++column)
        {
            INFO(L"[%f] ", ptr_array_inputs_received[column *
    rows_length_received + row]);
        }

        INFO(L"");
    }

    INFO(L"Output %ux%u: " , columns_length_received,
    rows_length_received); for(size_t column(0u), row(0u); row !=
    columns_length_received; ++row)
    {
        for(column = 0u; column != rows_length_received; ++column)
        {
            INFO(L"[%f] ", ptr_array_outputs_received[column *
    columns_length_received + row]);
        }

        INFO(L"");
    }
    */

    /*
    // Check error.
    size_t tmp_count_error = 0u;
    for(size_t row(0u),
                           column(0u); column != columns_length_received;
    ++column)
    {
        for(row = 0u; row != rows_length_received; ++row)
        {
            //INFO(L"Output[%u](%f) != Input[%u](%f)" ,
            //                        row * columns_length_received + column,
            //                        ptr_array_outputs_received[row *
    columns_length_received + column],
            //                        column * rows_length_received + row,
            //                        ptr_array_inputs_received[column *
    rows_length_received + row]);

            if(ptr_array_outputs_received[row * columns_length_received +
    column] != ptr_array_inputs_received[column * rows_length_received + row])
            { ++tmp_count_error; }
        }
    }

    INFO(L"sequentialMatrix %ux%u Total error: %u" ,
    rows_length_received, columns_length_received, tmp_count_error);
    // |END| Check error. |END|
    */
  }
}

template <typename T>
__device__ void Transpose(
    size_t const size_received, size_t const columns_length_received,
    size_t const rows_length_received, T *const ptr_array_outputs_received,
    T const *const ptr_array_inputs_received,
    struct dim3 const *const ptr_dimension_grid_recieved,
    struct dim3 const *const ptr_dimension_block_recieved) {
  if (rows_length_received == columns_length_received) {
    Transpose::Transpose_Square(
        size_received, rows_length_received, ptr_array_outputs_received,
        ptr_array_inputs_received, ptr_dimension_grid_recieved,
        ptr_dimension_block_recieved);
  } else {
    Transpose::Transpose_Rectangular(
        size_received, rows_length_received, columns_length_received,
        ptr_array_outputs_received, ptr_array_inputs_received,
        ptr_dimension_grid_recieved, ptr_dimension_block_recieved);
  }
}
}  // namespace Transpose