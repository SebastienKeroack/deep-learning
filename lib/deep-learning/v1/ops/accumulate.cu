#include "hip/hip_runtime.h"
﻿/* Copyright 2016, 2019 Sébastien Kéroack. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
==============================================================================*/

namespace DL::v1::Math {
template <typename T>
__global__ void kernel__Accumulate_X_X_1D(
    T *const ptr_array_outputs_received,
    T const *const ptr_array_inputs_received) {
  size_t const tmp_thread_global_index(blockIdx.x * blockDim.x + threadIdx.x);

  ptr_array_outputs_received[tmp_thread_global_index] +=
      ptr_array_inputs_received[tmp_thread_global_index];
}

template <typename T>
__global__ void kernel__Accumulate_X_X_1D(
    size_t const size_received, T *const ptr_array_outputs_received,
    T const *const ptr_array_inputs_received) {
  size_t const tmp_thread_global_index(blockIdx.x * blockDim.x + threadIdx.x);

  if (tmp_thread_global_index < size_received) {
    ptr_array_outputs_received[tmp_thread_global_index] +=
        ptr_array_inputs_received[tmp_thread_global_index];
  }
}

template <typename T>
__global__ void kernel_while__Accumulate_X_X_1D(
    size_t const size_received, T *const ptr_array_outputs_received,
    T const *const ptr_array_inputs_received) {
  size_t const tmp_grid_stride(gridDim.x * blockDim.x);
  size_t tmp_thread_global_index(blockIdx.x * blockDim.x + threadIdx.x);

  do {
    ptr_array_outputs_received[tmp_thread_global_index] +=
        ptr_array_inputs_received[tmp_thread_global_index];

    tmp_thread_global_index += tmp_grid_stride;
  } while (tmp_thread_global_index < size_received);
}

template <typename T>
__device__ void Accumulate_X_X_1D(
    size_t const size_received, T *ptr_array_outputs_received,
    T const *ptr_array_inputs_received,
    struct dim3 const *const ptr_dimension_grid_received,
    struct dim3 const *const ptr_dimension_block_received) {
  if (USE_PARALLEL && size_received >= warpSize) {
    LAUNCH_KERNEL_POINTER_1D(Accumulate_X_X_1D<T>, ptr_dimension_grid_received,
                             ptr_dimension_block_received, 0_UZ, size_received,
                             ptr_array_outputs_received,
                             ptr_array_inputs_received)
  } else {
    for (T const *const ptr_output_end(ptr_array_outputs_received +
                                       size_received);
         ptr_array_outputs_received != ptr_output_end;
         ++ptr_array_outputs_received, ++ptr_array_inputs_received) {
      *ptr_array_outputs_received += *ptr_array_inputs_received;
    }
  }
}

}  // namespace Accumulate